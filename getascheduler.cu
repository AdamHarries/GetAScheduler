#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

__global__ void run(void)
{
	int cid = threadIdx.x;
	int val = 0;
	while(val<(cid+10)){
		// do some "work" so the loop can't be compiled away
		val++;
		if(val == cid){
			val = 0;
		}
	}
}
int main(int argc, char** argv) 
{
	for(;;){
		run<<<1024,1024>>>();
		hipError_t err = hipDeviceSynchronize();
	}
	return 0;
}